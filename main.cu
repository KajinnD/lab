#include "hip/hip_runtime.h"
#include "prime.cpp"

int main(int argc, char *argv[]) {
   ULONGLONG n = atoi(argv[1]);
   ULONGLONG * liste = (ULONGLONG*) calloc(sqrt(n), sizeof(ULONGLONG));
   //searchPrimesCPU(n, liste);
   v0_factoCPU(n, liste);

   // AFFICHAGE DE LA DECOMPOSITION
   ULONGLONG i=0;
   while(liste[i] != 0) {
     printf("%llu\n", liste[i]);
     i++;
   }
   
   // TEST SI IL EST PREMIER
   if (isPrimeCPU(n)) {
     printf("true\n");
   } else {
     printf("false\n");
   }

   return 0;
}
