#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

#define ULONGLONG unsigned long long int

struct cell {
  ULONGLONG nbPrime;
  ULONGLONG exposant;
};

void completeInData(ULONGLONG n, ULONGLONG * inData) {
  ULONGLONG j = 1;
  inData[0] = 2;

  for (ULONGLONG i = 3; i < sqrt(n); i = i + 2) {
    inData[j] = i;
    j++;
  }
}

bool isPrimeCPU(const ULONGLONG N){
  if (N == 2) {
    return true;
  } else if (N%2 == 0) {
    return false;
  }

   for(int i=3 ; i*i < N ; i = i + 2){
      if (N % i == 0) {
         return false;
      }
   }
   return true;
}

void searchPrimesCPU(const ULONGLONG N, ULONGLONG * listPrime){
   ULONGLONG j = 0;
   listPrime[j] = 2;
   j++;

   // PAS SUR POUR LE i*i < N, peut être qu'il faut tous les prendre et alors i < N
   for (ULONGLONG i=3; i < N; i = i + 2 ){
     if (isPrimeCPU(i)) {
       listPrime[j] = i;
       j++;
     }
   }
}

void v0_factoCPU(ULONGLONG N, ULONGLONG * listFinale){
    ULONGLONG * listeP = (ULONGLONG*) calloc(N/2, sizeof(ULONGLONG));
    searchPrimesCPU(N, listeP);


    ULONGLONG tmp = N;
    ULONGLONG i = 0;
    ULONGLONG j = 0;

    if (isPrimeCPU(N)) {
      listFinale[0] = N;
    }
    while(listeP[i] != 0) {         //TANT QUE LA DECOMPOSITION EST PAS FINI
      if (tmp%listeP[i] == 0) {     //SI IL EST DIVISIBLE PAR LE NBP DE LA LISTE
        tmp = tmp/listeP[i];        // ON DIVISE TMP
        listFinale[j] = listeP[i];  //ON AJOUTE LE NBP DIVISEUR A LA LISTE listFinale
        j++;
      } else {                      // SINON ON INCREMENTE I POUR PASSER AU NBP SUIVANT DANS LA LISTE
        i++;
      }
    }
}

__global__ void v0_isPrimeGPU(ULONGLONG * inData, ULONGLONG * N, bool * isPrime) {
  if (isPrime){
     ULONGLONG tid = blockIdx.x * blockDim.x + threadIdx.x;

     while (tid < *N) {
       if (inData[tid] != 0) {
          if (*N % inData[tid] == 0) {
              *isPrime = false;
          }
       }
       tid += blockDim.x * gridDim.x;
     }
  }
}

__global__ void searchPrimesGPU(ULONGLONG * inData, bool * listePrime, ULONGLONG * N, ULONGLONG * outData) {

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  //pb avec le j
  __shared__ ULONGLONG * j;

  __syncthreads();

  while (tid < *N) {
    //SI LE CHIFFRE COURANT EST UN PRIME
      if (listePrime[tid]) {
          //cell prime;
          //prime.nbPrime = inData[tid];
          outData[*j] = inData[tid];
          //atomicAdd(*j, (ULONGLONG)1);
      }
      tid += blockDim.x * gridDim.x;
  }
}

__global__ void factoGPU(bool * listePrime, ULONGLONG * N, ULONGLONG * outData){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  ULONGLONG i = 0;
  ULONGLONG j = 0;

//SYNCH_THREAD ???
  while(tid < *N){
    if(*N % listePrime[i] == 0) {     //SI IL EST DIVISIBLE PAR LE NBP DE LA LISTE
       *N = *N/listePrime[i];          // ON DIVISE TMP
       outData[j] = listePrime[i];    //ON AJOUTE LE NBP DIVISEUR A LA LISTE listFinale
       j++;
    } else {                         // SINON ON INCREMENTE I POUR PASSER AU NBP SUIVANT DANS LA LISTE
       i++;
    }
    tid += blockDim.x * gridDim.x;
  }
}

int main(int argc, char *argv[]) {
   ULONGLONG n = atoi(argv[1]);

   ULONGLONG * inData = (ULONGLONG*) calloc(sqrt(n), sizeof(ULONGLONG));
   bool isPrime = true;
   bool *listPrime = (bool*) calloc(sqrt(n), sizeof(bool));
   ULONGLONG * outData = (ULONGLONG*) calloc(sqrt(n), sizeof(ULONGLONG));


   ULONGLONG j = 1;
   //inData[0] = 2;
   //for (ULONGLONG i = 3; i < sqrt(n); i = i + 2) {
  //   inData[j] = i;
  //   j++;
   //}

   // Liste on Device
   ULONGLONG *dev_inData;
   ULONGLONG *dev_n;
   bool *dev_isPrime;
   ULONGLONG * dev_outData;


   // Allocate memory on Device
   hipMalloc(&dev_inData, sqrt(n) * sizeof(ULONGLONG));
   hipMalloc(&dev_n, sizeof(ULONGLONG));
   hipMalloc(&dev_outData, sqrt(n) * sizeof(ULONGLONG));
   hipMalloc(&dev_isPrime, sizeof(bool));
   // Copy from Host to Device
	 hipMemcpy(dev_n, &n, sizeof(ULONGLONG), hipMemcpyHostToDevice);
   //hipMemcpy(dev_inData, inData, sqrt(n) * sizeof(ULONGLONG), hipMemcpyHostToDevice);
   //hipMemcpy(dev_isPrime, &isPrime, sizeof(bool), hipMemcpyHostToDevice);

   inData[0] = 2;
   for (ULONGLONG i = 3; i < sqrt(n); i = i + 2) {
      inData[j] = i;
      printf("%llu\n", *n%i);
      hipMemcpy(dev_inData, inData, sqrt(n) * sizeof(ULONGLONG), hipMemcpyHostToDevice);
      hipMemcpy(dev_isPrime, &isPrime, sizeof(bool), hipMemcpyHostToDevice);
      v0_isPrimeGPU<<<n+1/512, 512>>>(dev_inData, dev_n, dev_isPrime);
      hipMemcpy(&isPrime, dev_isPrime, sizeof(bool), hipMemcpyDeviceToHost);
      listPrime[j] = isPrime;
      j++;
      hipFree(dev_inData);
      hipFree(dev_isPrime);
      if (isPrime) {
        printf("true\n");
      } else {
        printf("false\n");
      }
    }

  /*hipMemcpy(dev_isPrime, listPrime, sizeof(ULONGLONG), hipMemcpyHostToDevice);
    searchPrimesGPU<<<n+1/512, 512>>>(dev_inData, dev_isPrime, dev_n, dev_outData);
    hipMemcpy(&outData, dev_outData, sqrt(n) * sizeof(ULONGLONG), hipMemcpyDeviceToHost);
*/
    // Launch kernel
    //v0_isPrimeGPU<<<10, 32>>>(dev_inData, dev_n, dev_isPrime);

		// Copy from Device to Host
    //hipMemcpy(&isPrime, dev_isPrime, sizeof(bool), hipMemcpyDeviceToHost);

    if (isPrime) {
      printf("true\n");
    } else {
      printf("false\n");
    }

    ULONGLONG i=0;
    while(outData[i] != 0) {
      printf("%llu\n", outData[i]);
      i++;
    }

		// Free memory on Device
		hipFree(dev_inData);
		hipFree(dev_n);
    hipFree(dev_isPrime);

   return 0;
}
